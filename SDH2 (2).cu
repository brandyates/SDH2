#include "hip/hip_runtime.h"
/* ==================================================================
	// histogram privatization + tiling on shared mem
   ==================================================================
*/

#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <sys/time.h>
#include <stdint.h>
#include "timing.cuh"


#define BOX_SIZE	23000 /* size of the data box on one dimension            */
#define BUCKET_TYPE unsigned long long // data type of a bucket

/* descriptors for single atom in the tree */
typedef struct atomdesc {
	float x_pos;
	float y_pos;
	float z_pos;
} atom;

typedef struct hist_entry{
	//float min;
	//float max;
	long long d_cnt;   /* need a long long type as the count might be huge */
} bucket;


BUCKET_TYPE * histogram;	/* list of all buckets in the histogram   */
unsigned PDH_acnt;		/* total number of data points            */
int num_buckets;		/* total number of buckets in the histogram */
float   PDH_res;		/* value of w                             */
atom * atom_list;		/* list of all data points                */

/* These are for an old way of tracking time */
struct timezone Idunno;	
struct timeval startTime, endTime;


/* 
	distance of two points in the atom_list 
*/
float p2p_distance(int ind1, int ind2) {
	
	double x1 = atom_list[ind1].x_pos;
	double x2 = atom_list[ind2].x_pos;
	double y1 = atom_list[ind1].y_pos;
	double y2 = atom_list[ind2].y_pos;
	double z1 = atom_list[ind1].z_pos;
	double z2 = atom_list[ind2].z_pos;
		
	return sqrt((x1 - x2)*(x1-x2) + (y1 - y2)*(y1 - y2) + (z1 - z2)*(z1 - z2));
}


/* 
	brute-force SDH solution in a single CPU thread 
*/
int PDH_baseline() {
	int i, j, h_pos;
	float dist;
	
	for(i = 0; i < PDH_acnt; i++) {
		for(j = i+1; j < PDH_acnt; j++) {
			dist = p2p_distance(i,j);
			h_pos = (int) (dist / PDH_res);
			histogram[h_pos]++;
		} 
	}
	return 0;
}

/* 
	set a checkpoint and show the (natural) running time in seconds 
*/
float report_running_time() {
	long sec_diff, usec_diff;
	gettimeofday(&endTime, &Idunno);
	sec_diff = endTime.tv_sec - startTime.tv_sec;
	usec_diff= endTime.tv_usec-startTime.tv_usec;
	if(usec_diff < 0) {
		sec_diff --;
		usec_diff += 1000000;
	}
	printf("Running time for CPU version: %ld.%06ld\n", sec_diff, usec_diff);
	return (float)(sec_diff*1.0 + usec_diff/1000000.0);
}


/* 
	print the counts in all buckets of the histogram 
*/
void output_histogram(BUCKET_TYPE * _histogram){
	int i; 
	long long total_cnt = 0;
	for(i=0; i< num_buckets; i++) {
		if(i%5 == 0) /* we print 5 buckets in a row */
			printf("\n%02d: ", i);
		int tmp = (int)_histogram[i];
		printf("%15lld ", (long long)tmp);
		total_cnt += _histogram[i];
	  	/* we also want to make sure the total distance count is correct */
		if(i == num_buckets - 1)	
			printf("\n T:%lld \n", total_cnt);
		else printf("| ");
	}
}

/* histogram differences */
BUCKET_TYPE * hist_diff(BUCKET_TYPE * hist1, BUCKET_TYPE * hist2){
	BUCKET_TYPE * diff = (BUCKET_TYPE*)malloc(num_buckets*sizeof(BUCKET_TYPE));
	for (int i=0; i<num_buckets; i++){
		diff[i] = hist1[i] - hist2[i];
	}
	return diff;
}

/* ---------------------------------------------------- GPU CODE -------------------------------------------------- */

__global__ void kernel(float* d_x, float* d_y, float* d_z, int n_atoms, BUCKET_TYPE * d_hist, float PDH_res){
	
}


/* ---------------------------------------------------- MAIN -------------------------------------------------- */
int main(int argc, char **argv)
{
	int i;

	PDH_acnt = atoi(argv[1]);
	PDH_res	 = (float)atof(argv[2]);
	int blockDim = BLOCKDIM;


	if (argc > 3) blockDim = atoi(argv[3]);

	num_buckets = (int)(BOX_SIZE * 1.732 / PDH_res) + 1;
	histogram = (BUCKET_TYPE *)malloc(sizeof(BUCKET_TYPE)*num_buckets);

	atom_list = (atom *)malloc(sizeof(atom)*PDH_acnt);
	float* h_x = (float*)malloc(PDH_acnt*sizeof(float));
	float* h_y = (float*)malloc(PDH_acnt*sizeof(float));
	float* h_z = (float*)malloc(PDH_acnt*sizeof(float));

	srand(1);
	/* generate data following a uniform distribution */
	for(i = 0;  i < PDH_acnt; i++) {
		h_x[i] = ((float)(rand()) / RAND_MAX) * BOX_SIZE;
		h_y[i] = ((float)(rand()) / RAND_MAX) * BOX_SIZE;
		h_z[i] = ((float)(rand()) / RAND_MAX) * BOX_SIZE;
		atom_list[i].x_pos = h_x[i];
		atom_list[i].y_pos = h_y[i];
		atom_list[i].z_pos = h_z[i];
	}

	// /* call CPU single thread version to compute the histogram */
	// /* start counting time */
	// TIMING_START();
	
	// // PDH_baseline();
	// PDH_baseline();
	
	// /* check the total running time */ 
	// TIMING_STOP();
	
	// /* print out the histogram */
	// printf("\nCPU results: \n");
	// output_histogram(histogram);
	// TIMING_PRINT();

	/* ---------------------------- GPU version ---------------------------- */
	// copy data to GPU 
	float *d_x, *d_y, *d_z;
	hipMalloc((void**)&d_x, PDH_acnt*sizeof(float));
	hipMalloc((void**)&d_y, PDH_acnt*sizeof(float));
	hipMalloc((void**)&d_z, PDH_acnt*sizeof(float));
	hipMemcpy(d_x, h_x, PDH_acnt*sizeof(float), hipMemcpyHostToDevice);
	hipMemcpy(d_y, h_y, PDH_acnt*sizeof(float), hipMemcpyHostToDevice);
	hipMemcpy(d_z, h_z, PDH_acnt*sizeof(float), hipMemcpyHostToDevice);

	// create output space
	BUCKET_TYPE *d_hist;
	hipMalloc((void**)&d_hist, num_buckets*sizeof(BUCKET_TYPE));
	hipMemset(d_hist, 0, num_buckets*sizeof(BUCKET_TYPE));

	// dynamic shared mem
	size_t sharedMemSize = num_buckets*sizeof(unsigned);

	TIMING_START();
	kernel <<< ceil((float)PDH_acnt/blockDim) , blockDim, sharedMemSize >>> (d_x, d_y, d_z, PDH_acnt, d_hist, PDH_res);
	hipDeviceSynchronize();
	TIMING_STOP();

	// copy data back to host
	BUCKET_TYPE * histogram_GPU = (BUCKET_TYPE*)malloc(num_buckets*sizeof(BUCKET_TYPE));
	hipMemcpy(histogram_GPU, d_hist, num_buckets*sizeof(BUCKET_TYPE), hipMemcpyDeviceToHost);
	// print out
	printf("\nGPU results: \n");
	output_histogram(histogram_GPU);
	TIMING_PRINT();

	// check diff
	BUCKET_TYPE * diff = hist_diff(histogram_GPU, histogram);
	printf("\nDIFFERENCES: \n");
	output_histogram(diff);
	
	return 0;
}

